#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"

//texture samplers
rtTextureSampler<uchar4, 2, hipReadModeElementType> input_texture;

//output buffers
rtBuffer<uchar4, 2>   output_buffer;

//optix tracked variables
rtDeclareVariable(uint2,	launch_index,	rtLaunchIndex, );
rtDeclareVariable(uint2,	launch_dim,		rtLaunchDim, );

RT_PROGRAM void draw_texture()
{
	float2 texcoord = make_float2( launch_index ) / make_float2( launch_dim );

	output_buffer[launch_index] = rgba_to_bgra( tex2D( input_texture, texcoord.x, texcoord.y ) );
}
