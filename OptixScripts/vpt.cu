#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "scene.h"
#include "lights.h"
#include "helpers.h"
#include "path_tracer.h"
#include "camera.h"
//#include "random.h"

using namespace optix;

struct PerRayData_pathtrace
{
	float3 result;
	float3 radiance;
	float3 attenuation;
	float3 origin;
	float3 direction;
	unsigned int seed;
	int depth;
	int countEmitted;
	int done;
	int inside;
};


// For camera
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtBuffer<ParallelogramLight>     plights;

//ray types
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  max_depth, , );

//output buffers
rtBuffer<float4, 2>              output_buffer;

//rays
rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

//optix tracked data
rtDeclareVariable(uint2,		launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2,		launch_dim,   rtLaunchDim, );

__device__ __inline__ float sample_distance(const float e, const float sig_t)
{
	return -log(e) / sig_t;
}
__device__ __inline__ float  sampleSegment(float epsilon, float sigma, float smax) {
	return -logf(1.0f - epsilon * (1.0f - expf(-sigma * smax))) / sigma;
}
__device__ __inline__ float phase(const float3& wo, const float3& wi)
{
	return 1.0f / (4.0f*M_PIf);
}

__device__ __inline__ float3 sample_HG(float g, float e1, float e2) {
	//double s=2.0*e1-1.0, f = (1.0-g*g)/(1.0+g*s), cost = 0.5*(1.0/g)*(1.0+g*g-f*f), sint = sqrt(1.0-cost*cost);
	float s = 1.0f - 2.0f*e1, cost = (s + 2.0f*g*g*g * (-1.0f + e1) * e1 + g*g*s + 2.0f*g*(1.0f - e1 + e1*e1)) / ((1.0f + g*s)*(1.0f + g*s)), sint = sqrtf(1.0f - cost*cost);
	return make_float3(cosf(2.0f * M_PIf * e2) * sint, sinf(2.0f * M_PIf * e2) * sint, cost);
}

// For miss program
rtDeclareVariable(float3,       bg_color, , );

//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

RT_PROGRAM void pathtrace_camera()
{
	size_t2 screen = output_buffer.size();

	float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

	float2 jitter_scale = inv_screen / sqrt_num_samples;
	unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
	float3 result = make_float3(0.0f);

	unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
	do {
		unsigned int x = samples_per_pixel%sqrt_num_samples;
		unsigned int y = samples_per_pixel/sqrt_num_samples;
		float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
		float2 d = pixel + jitter*jitter_scale;
		float3 ray_origin = eye;
		float3 ray_direction = normalize(d.x*U + d.y*V + W);

		//float camPdf;
		//GenerateRay(jitter, ray_origin, ray_direction, &camPdf);

		PerRayData_pathtrace prd;
		prd.result = make_float3(0.f);
		prd.attenuation = make_float3(1.f);
		prd.countEmitted = true;
		prd.done = false;
		prd.inside = false;
		prd.seed = seed;
		prd.depth = 0;

		for(;;) {
			Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, prd);
			if(prd.done) {
				prd.result += prd.radiance * prd.attenuation;
				break;
			}

			// RR
			if(prd.depth >= rr_begin_depth){
				//break;
				float pcont = fmaxf(prd.attenuation);
				if(rnd(prd.seed) >= pcont || prd.depth > max_depth )
					break;
				prd.attenuation /= pcont;
			}
			prd.depth++;
			prd.result += prd.radiance * prd.attenuation;
			ray_origin = prd.origin;
			ray_direction = prd.direction;
		} // eye ray

		result += prd.result;
		seed = prd.seed;
	} while (--samples_per_pixel);

	float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

	if (frame_number > 1)
	{
		float a = 1.0f / (float)frame_number;
		float b = ((float)frame_number - 1.0f) * a;
		float3 old_color = make_float3(output_buffer[launch_index]);
		output_buffer[launch_index] = make_float4(a * pixel_color + b * old_color, 0.0f);
	}
	else
	{
		output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
	}
}

rtDeclareVariable(float3,        emission_color, , );
rtDeclareVariable(float3,        diffuse_color, , );

RT_PROGRAM void diffuse()
{
	//light surface
	if( length( emission_color ) > 0.0f )
	{
		current_prd.radiance = current_prd.countEmitted? emission_color : make_float3(0.f);
		current_prd.done = true;
		return;
	}

	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	current_prd.origin = hitpoint;

	float z1=rnd(current_prd.seed);
	float z2=rnd(current_prd.seed);
	float3 p;
	cosine_sample_hemisphere(z1, z2, p);
	float3 v1, v2;
	createONB(ffnormal, v1, v2);
	current_prd.direction = v1 * p.x + v2 * p.y + ffnormal * p.z;
	float3 normal_color = (normalize(world_shading_normal)*0.5f + 0.5f)*0.9;
	current_prd.attenuation = current_prd.attenuation * diffuse_color; // use the diffuse_color as the diffuse response
	current_prd.countEmitted = false;

	// Compute direct light...
	// Or shoot one...
	unsigned int num_lights = plights.size();
	float3 result = make_float3(0.0f);

	for(int i = 0; i < num_lights; ++i) {
		ParallelogramLight light = plights[i];
		float z1 = rnd(current_prd.seed);
		float z2 = rnd(current_prd.seed);
		float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		float Ldist = length(light_pos - hitpoint);
		float3 L = normalize(light_pos - hitpoint);
		float nDl = dot( ffnormal, L );
		float LnDl = dot( light.normal, L );
		float A = length(cross(light.v1, light.v2));

		// cast shadow ray
		if ( nDl > 0.0f && LnDl > 0.0f ) {
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = false;
			Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_object, shadow_ray, shadow_prd);

			if(!shadow_prd.inShadow){
				float weight = nDl * LnDl * A / (M_PIf*Ldist*Ldist);
				result += light.emission * weight;
			}
		}
	}

	current_prd.radiance = result;
}

rtDeclareVariable(float3,        specular_color, , );
rtDeclareVariable(float,         index_of_refraction, , );

RT_PROGRAM void glass_refract()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	current_prd.origin = hitpoint;
	current_prd.countEmitted = true;
	float iof;
	if (current_prd.inside) {
		// Shoot outgoing ray
		iof = 1.0f/index_of_refraction;
	} else {
		iof = index_of_refraction;
	}
	refract(current_prd.direction, ray.direction, ffnormal, iof);
	//prd.direction = reflect(ray.direction, ffnormal);

	if (current_prd.inside) {
		// Compute Beer's law
		current_prd.attenuation = current_prd.attenuation * powf(specular_color, t_hit);
	}
	current_prd.inside = !current_prd.inside;

	current_prd.radiance = make_float3(0.0f);
}

RT_PROGRAM void specular()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	current_prd.origin = hitpoint;
	current_prd.countEmitted = true;
	current_prd.radiance = make_float3(0.0f);

	// specular reflection
	current_prd.direction = reflect(ray.direction, ffnormal);
	current_prd.attenuation = current_prd.attenuation * diffuse_color;
}

RT_PROGRAM void diffuse_volume()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	current_prd.origin = hitpoint;

	float z1=rnd(current_prd.seed);
	float z2=rnd(current_prd.seed);
	float3 p;
	cosine_sample_hemisphere(z1, z2, p);
	float3 v1, v2;
	createONB(ffnormal, v1, v2);
	current_prd.direction = v1 * p.x + v2 * p.y + ffnormal * p.z;
	float3 normal_color = (normalize(world_shading_normal)*0.5f + 0.5f)*0.9;
	current_prd.attenuation = current_prd.attenuation * diffuse_color; // use the diffuse_color as the diffuse response
	current_prd.countEmitted = false;

	// Compute direct light...
	// Or shoot one...
	unsigned int num_lights = plights.size();
	float3 result = make_float3(0.0f);

	for(int i = 0; i < num_lights; ++i) {
		ParallelogramLight light = plights[i];
		float z1 = rnd(current_prd.seed);
		float z2 = rnd(current_prd.seed);
		float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		float Ldist = length(light_pos - hitpoint);
		float3 L = normalize(light_pos - hitpoint);
		float nDl = dot( ffnormal, L );
		float LnDl = dot( light.normal, L );
		float A = length(cross(light.v1, light.v2));

		// cast shadow ray
		if ( nDl > 0.0f && LnDl > 0.0f ) {
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = false;
			Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_object, shadow_ray, shadow_prd);

			if(!shadow_prd.inShadow){
				float weight = nDl * LnDl * A / (M_PIf*Ldist*Ldist);
				result += light.emission * weight;
			}
		}
	}

	current_prd.radiance = result;
}

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
	current_prd_shadow.inShadow = true;
	current_prd_shadow.attenuation = make_float3(0);
	rtTerminateRay();
}

RT_PROGRAM void vol_shadow()
{
	current_prd_shadow.inShadow = false;
	current_prd_shadow.inside = !current_prd_shadow.inside;
	if (!current_prd_shadow.inside)
	{
		current_prd.attenuation = expf(-t_hit*0.1f)*diffuse_color;
	}
	rtIgnoreIntersection();
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(bad_color, 0.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
	current_prd.radiance = bg_color;
	current_prd.done = true;
}

